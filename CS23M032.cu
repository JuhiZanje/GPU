#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here

__device__ int gcm(int a,int b){
    if(b==0)    return a;
    return gcm(b,a%b);
}

__global__ void setDHTempScore(int *DHP,int *tempDHP,int *Dscore,int H){
    int id=threadIdx.x;
    DHP[id]=H;
    tempDHP[id]=H;
    Dscore[id]=0;
}

__global__ void initGCM(int *gcmStore,int T){
    int ind=blockIdx.x * T + threadIdx.x;
    gcmStore[ind]=0;
}

__global__ void updateDH(int *DHP,int *tempDHP, int *countActiveTank,int T,int round){
    if((round%T)!=0){
        int id=threadIdx.x;
        if(tempDHP[id]<=0 && DHP[id]>0)
            atomicAdd(countActiveTank,-1);        
        DHP[id]=tempDHP[id];
    }                    
}

__global__ void roundKernel(int round,int T,int *DHP,int *tempDHP,int* Dxcoord,int* Dycoord,int* Dscore,int *countActiveTank,int *gcmStore){
    __shared__ int tankToShoot;
    __shared__ volatile int lock;
    __shared__ volatile int minK;
    if((round%T)!=0){
        int j=threadIdx.x;
        int id=blockIdx.x;

        if(j==0){
            minK=INT_MAX;
            tankToShoot=-1;
            lock=0;
        }

        __syncthreads();

        if(DHP[id]>0){
            int hitDir=(id+round)%T;
            int x=Dxcoord[id],y=Dycoord[id];
            int diffx=Dxcoord[hitDir]-x;
            int diffy=Dycoord[hitDir]-y;
            int gcmVal,ind;
            ind=hitDir * T + id;
            if(gcmStore[ind]!=0){
                gcmVal=gcmStore[ind];
            }else{
                gcmVal=abs(gcm(diffx,diffy));
                gcmStore[ind]=gcmVal;
            }        
            int dirx=diffx/gcmVal;
            int diry=diffy/gcmVal;

            if(DHP[j]>0 && id!=j ){
                diffx=Dxcoord[j]-x;
                diffy=Dycoord[j]-y;
                ind=j * T + id;
                if(gcmStore[ind]!=0){
                    gcmVal=gcmStore[ind];
                }else{
                    gcmVal=abs(gcm(diffx,diffy)); 
                    gcmStore[ind]=gcmVal;
                }                

                int dirxN=diffx/gcmVal;
                int diryN=diffy/gcmVal;                

                if(dirxN==dirx && diry==diryN){
                    int k;
                    if(dirx!=0)
                        k=diffx/dirx;
                    else
                        k=diffy/diry;
                    int old;
                    if(k<minK){
                        for(int i=0;i<32;i++){
                            if(j%32==i){
                                do{
                                    old=atomicCAS((int *)&lock,0,1);
                                    if(old==0){
                                        if(k<minK){ 
                                            minK=k;
                                            tankToShoot=j;
                                        }
                                        lock=0;
                                    }
                                }while(old!=0);
                            }
                        }               
                    
                    }                
                }
            }            

        }
        __syncthreads();    
        if(j==0){
            if(tankToShoot!=-1){            
                atomicAdd(&tempDHP[tankToShoot],-1);            
                Dscore[id]++;
            }
        } 
    }
         
}


//***********************************************


int main(int argc,char **argv)
{    
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int id=0;id<T;id++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[id] );
      fscanf( inputfilepointer, "%d", &ycoord[id] );
    }
		

    auto start = chrono::high_resolution_clock::now();


    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *countActiveTank;
    int *Dxcoord,*Dycoord,*Dscore;
    hipMalloc(&Dxcoord , T * sizeof (int)) ;
	hipMemcpy(Dxcoord,xcoord,T * sizeof (int),hipMemcpyHostToDevice);

    hipMalloc(&Dycoord , T * sizeof (int)) ;
	hipMemcpy(Dycoord,ycoord,T * sizeof (int),hipMemcpyHostToDevice);

    hipMalloc(&Dscore , T * sizeof (int)) ;
	// cudaMemcpy(Dscore,score,T * sizeof (int),cudaMemcpyHostToDevice);

    int *DH;
    hipMalloc(&DH , T * sizeof (int));

    int *tempDH;
    hipMalloc(&tempDH , T * sizeof (int));
    setDHTempScore<<<1,T>>>(DH,tempDH,Dscore,H);

    int *gcmStore;
    hipMalloc(&gcmStore , T * T* sizeof (int));   
    initGCM<<<T,T>>>(gcmStore,T);    

    hipHostAlloc(&countActiveTank,sizeof(int),0);
    *countActiveTank=T;
    int round=1; 
    
    while(*countActiveTank>1){  
        roundKernel<<<T,T>>>(round,T,DH,tempDH,Dxcoord,Dycoord,Dscore,countActiveTank,gcmStore);
        updateDH<<<1,T>>>(DH,tempDH,countActiveTank,T,round);              
        hipDeviceSynchronize();        
        round++;       
    }   

	hipMemcpy(score,Dscore,T * sizeof (int),hipMemcpyDeviceToHost);   
    
    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int id=0;id<T;id++)
    {
        fprintf( outputfilepointer, "%d\n", score[id]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}